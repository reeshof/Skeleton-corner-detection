#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "include/skelft.h"
#include <stdio.h>



// Parameters for CUDA kernel executions; more or less optimized for a 1024x1024 image.
#define BLOCKX		16
#define BLOCKY		16
#define BLOCKSIZE	64
#define TILE_DIM	32
#define BLOCK_ROWS	16



/****** Global Variables *******/
const int NB = 7;						// Nr buffers we use and store in the entire framework
short2 **pbaTextures;					// Work buffers used to compute and store resident images
//	0: work buffer
//	1: FT
//	2: thresholded DT
//	3: thresholded skeleton
//	4: topology analysis
//  5: work buffer for topology
//  6: skeleton FT
//

float*			pbaTexSiteParam;		// Stores boundary parameterization (>0: boundary-code; =0: non-boundary pixel)
int					pbaTexSize;					// Texture size (squared) actually used in all computations
int					floodBand  = 4,			// Various FT computation parameters; defaults are good for an 1024x1024 image.
						maurerBand = 4,
						colorBand  = 4;

texture<short2> pbaTexColor;			// 2D textures (bound to various buffers defined above as needed)
texture<short2> pbaTexColor2;			//
texture<short2> pbaTexLinks;
texture<float>  pbaTexParam;			// 1D site parameterization texture (bound to pbaTexSiteParam)
texture<unsigned char> pbaTexGray;	// 2D texture of unsigned char values, e.g. the binary skeleton

#if __CUDA_ARCH__ < 110					// We cannot use atomic intrinsics on SM10 or below. Thus, we define these as nop.
//#define atomicInc(a,b) 0				// The default will be that some code e.g. endpoint detection will thus not do anything.
#endif


__device__ unsigned int topo_gc			= 0;		//Used for pixel-array-creation (for skeleton endpoints and skeletons)
__device__ unsigned int topo_gc_last	= 0;

#define X 255

__constant__ const																					//REMARK: put following constants (for kernelTopology) in CUDA constant-memory, as this gives a huge speed difference
unsigned char topo_patterns[][9] =		{ {0,0,0,							//These are the 3x3 templates that we use to detect skeleton endpoints
										   0,X,0,																//(with four 90-degree rotations for each)
										   0,X,0},
										  {0,0,0,
										   0,X,0,
										   0,0,X},
										  {0,0,0,
										   0,X,0,
										   0,X,X},
										  {0,0,0,
										   0,X,0,
										   X,X,0}
										};

#define topo_NPATTERNS  4																	//Number of patterns we try to match (for kernelTopology)
																													//REMARK: #define faster than __constant__

__constant__ const unsigned char topo_rot[][9] = { {0,1,2,3,4,5,6,7,8}, {2,5,8,1,4,7,0,3,6}, {8,7,6,5,4,3,2,1,0}, {6,3,0,7,4,1,8,5,2} };
																				//These encode the four 90-degree rotations of the patterns (for kernelTopology);


#include "include/skelftKernel.h"



// Initialize necessary memory (CPU/GPU sides)
// - textureSize: The max size of any image we will process until re-initialization
void skelft2DInitialization(int maxTexSize)
{
	  hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,0);																			// Query device properties, list something about them

    int pbaMemSize = maxTexSize * maxTexSize * sizeof(short2);								// A buffer has 2 shorts / pixel

    pbaTextures  = (short2 **) malloc(NB * sizeof(short2*));									// We will use NB buffers

	  for(int i=0;i<NB;++i)
       hipMalloc((void **) &pbaTextures[i], pbaMemSize);											// Allocate work buffer 'i'

    hipMalloc((void **) &pbaTexSiteParam, maxTexSize * maxTexSize * sizeof(float));		// Sites texture
}

// Deallocate all allocated memory
void skelft2DDeinitialization()
{
    for(int i=0;i<NB;++i) hipFree(pbaTextures[i]);
	hipFree(pbaTexSiteParam);
    free(pbaTextures);
}

__global__ void kernelSiteParamInit(short2* inputVoro, int size)							//Initialize the Voronoi textures from the sites' encoding texture (parameterization)
{																																							//REMARK: we interpret 'inputVoro' as a 2D texture, as it's much easier/faster like this
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

  if (tx<size && ty<size)																											//Careful not to go outside the image..
	{
	  int i = TOID(tx,ty,size);
	  float param = tex1Dfetch(pbaTexParam,i);																	//The sites-param has non-zero (parameter) values precisely on non-boundary points

	  short2& v = inputVoro[i];
	  v.x = v.y = MARKER;																												//Non-boundary points are marked as 0 in the parameterization. Here we will compute the FT.
	  if (param>0)																															//These are points which define the 'sites' to compute the FT/skeleton (thus, have FT==identity)
	  {																																					//We could use an if-then-else here, but it's faster with an if-then
	     v.x = tx; v.y = ty;
	  }
	}
}



void skelft2DInitializeInput(float* siteParam, int size)									// Copy input sites from CPU to GPU; Also set up site param initialization in pbaTextures[0]
{
    pbaTexSize = size;																										// Size of the actual texture being used in this run; can be smaller than the max-tex-size
																																					// which was used in skelft2DInitialization()

	hipMemcpy(pbaTexSiteParam, siteParam, pbaTexSize * pbaTexSize * sizeof(float), hipMemcpyHostToDevice);
																																					// Pass sites parameterization to CUDA.  Must be done before calling the initialization
																																					// kernel, since we use the sites-param as a texture in that kernel
	hipBindTexture(0, pbaTexParam, pbaTexSiteParam);												// Bind the sites-param as a 1D texture so we can quickly index it next
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

	kernelSiteParamInit<<<grid,block>>>(pbaTextures[0],pbaTexSize);					// Do the site param initialization. This sets up pbaTextures[0]
	hipUnbindTexture(pbaTexParam);																					// Done with the sites-param 1D texture
}





// In-place transpose a squared texture.
// Block orders are modified to optimize memory access.
// Point coordinates are also swapped.
void pba2DTranspose(short2 *texture)
{
    dim3 block(TILE_DIM, BLOCK_ROWS);
    dim3 grid(pbaTexSize / TILE_DIM, pbaTexSize / TILE_DIM);

    hipBindTexture(0, pbaTexColor, texture);
    kernelTranspose<<< grid, block >>>(texture, pbaTexSize);
    hipUnbindTexture(pbaTexColor);
}

// Phase 1 of PBA. m1 must divides texture size
void pba2DPhase1(int m1, short xm, short ym, short xM, short yM)
{
    dim3 block = dim3(BLOCKSIZE);
    dim3 grid = dim3(pbaTexSize / block.x, m1);

    // Flood vertically in their own bands
    hipBindTexture(0, pbaTexColor, pbaTextures[0]);
    kernelFloodDown<<< grid, block >>>(pbaTextures[1], pbaTexSize, pbaTexSize / m1);
    hipUnbindTexture(pbaTexColor);

    hipBindTexture(0, pbaTexColor, pbaTextures[1]);
    kernelFloodUp<<< grid, block >>>(pbaTextures[1], pbaTexSize, pbaTexSize / m1);

    // Passing information between bands
    grid = dim3(pbaTexSize / block.x, m1);
    kernelPropagateInterband<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m1);

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]);
    kernelUpdateVertical<<< grid, block >>>(pbaTextures[1], pbaTexSize, m1, pbaTexSize / m1);
    hipUnbindTexture(pbaTexLinks);
    hipUnbindTexture(pbaTexColor);
}

// Phase 2 of PBA. m2 must divides texture size
void pba2DPhase2(int m2)
{
    // Compute proximate points locally in each band
    dim3 block = dim3(BLOCKSIZE);
    dim3 grid = dim3(pbaTexSize / block.x, m2);
    hipBindTexture(0, pbaTexColor, pbaTextures[1]);
    kernelProximatePoints<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m2);

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]);
    kernelCreateForwardPointers<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m2);

    // Repeatly merging two bands into one
    for (int noBand = m2; noBand > 1; noBand /= 2) {
        grid = dim3(pbaTexSize / block.x, noBand / 2);
        kernelMergeBands<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / noBand);
    }

    // Replace the forward link with the X coordinate of the seed to remove
    // the need of looking at the other texture. We need it for coloring.
    grid = dim3(pbaTexSize / block.x, pbaTexSize);
    kernelDoubleToSingleList<<< grid, block >>>(pbaTextures[0], pbaTexSize);
    hipUnbindTexture(pbaTexLinks);
    hipUnbindTexture(pbaTexColor);
}

// Phase 3 of PBA. m3 must divides texture size
void pba2DPhase3(int m3)
{
    dim3 block = dim3(BLOCKSIZE / m3, m3);
    dim3 grid = dim3(pbaTexSize / block.x);
    hipBindTexture(0, pbaTexColor, pbaTextures[0]);
    kernelColor<<< grid, block >>>(pbaTextures[1], pbaTexSize);
    hipUnbindTexture(pbaTexColor);
}



void skel2DFTCompute(short xm, short ym, short xM, short yM, int floodBand, int maurerBand, int colorBand)
{
    pba2DPhase1(floodBand,xm,ym,xM,yM);										//Vertical sweep

    pba2DTranspose(pbaTextures[1]);											//

    pba2DPhase2(maurerBand);												//Horizontal coloring

    pba2DPhase3(colorBand);													//Row coloring

    pba2DTranspose(pbaTextures[1]);
}





__global__ void kernelThresholdDT(unsigned char* output, int size, float threshold2, short xm, short ym, short xM, short yM)
//Input:    pbaTexColor: closest-site-ids per pixel, i.e. FT
//Output:   output: thresholded DT
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if (tx>xm && ty>ym && tx<xM && ty<yM)									//careful not to index outside the image..
	{
  	  int    id     = TOID(tx, ty, size);
	  short2 voroid = tex1Dfetch(pbaTexColor,id);							//get the closest-site to tx,ty into voroid.x,.y
	  float  d2     = (tx-voroid.x)*(tx-voroid.x)+(ty-voroid.y)*(ty-voroid.y);
	  output[id]    = (d2<=threshold2);										//threshold DT into binary image
    }
}



__global__ void kernelDT(short* output, int size, float threshold2, short xm, short ym, short xM, short yM)
//Input:    pbaTexColor: closest-site-ids per pixel, i.e. FT
//Output:   output: DT
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if (tx>xm && ty>ym && tx<xM && ty<yM)									//careful not to index outside the image..
	{
  	  int    id     = TOID(tx, ty, size);
	    short2 voroid = tex1Dfetch(pbaTexColor,id);							//get the closest-site to tx,ty into voroid.x,.y
	    float  d2     = (tx-voroid.x)*(tx-voroid.x)+(ty-voroid.y)*(ty-voroid.y);
	    output[id]    = sqrtf(d2);											//save the Euclidean DT
  }
}





__global__ void kernelSkel(unsigned char* output, short xm, short ym,
						   short xM, short yM, short size, float threshold, float length,
						   float*  output_skelmaxImp, short2* output_skelmaxPos, const float SKELFT_FAR_AWAY, float* importanceDevice)
																			//Input:    pbaTexColor: closest-site-ids per pixel
																			//			    pbaTexParam: labels for sites (only valid at site locations)
{																			//Output:	  output: binary thresholded skeleton; serialized skeleton pixels
																			//WARNING: this kernel may sometimes create 2-pixel-thick branches.. Study the AFMM original code to see if this is correct.

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	int    id = TOID(tx, ty, size);

	//compare the current location with its own feature transform. For boundary points these are the same, thus onyl the boundary points are ignored.
	short2 curLocation { tx,ty };
	short2 ftLocation = tex1Dfetch(pbaTexColor, id);

	if (tx>xm && ty>ym && tx<xM-1 && ty<yM-1 && !(ftLocation.x == curLocation.x && ftLocation.y == curLocation.y))
	{
  	
	  //if (tex1Dfetch(pbaTexParam,id)==SKELFT_FAR_AWAY)						//compute only the foreground-skeleton (the background one is usually not interesting)
	  //{
	    int    Id     = id;
	    short2 voroid = tex1Dfetch(pbaTexColor,id);							  //get the closest-site to tx,ty into voroid.x,.y
		  float  d2     = sqrtf((tx-voroid.x)*(tx-voroid.x)+(ty-voroid.y)*(ty-voroid.y));	//get DT^2 of tx,ty

	    int    id2    = TOID(voroid.x,voroid.y,size);						//convert the site's coord to an index into pbaTexParam[], the site-label-texture
	    float  imp    = tex1Dfetch(pbaTexParam,id2);						//get the site's label

	           ++id;																						//TOID(tx+1,ty,size)
	           voroid = tex1Dfetch(pbaTexColor,id);							//
	           id2    = TOID(voroid.x,voroid.y,size);						//
	    float  imp_r  = tex1Dfetch(pbaTexParam,id2);						//

			id -= 2;																						//TOID(tx+1,ty,size)
			voroid = tex1Dfetch(pbaTexColor, id);							//
			id2 = TOID(voroid.x, voroid.y, size);						//
		float  imp_r2 = tex1Dfetch(pbaTexParam, id2);						//

	           id     += size+1;																//TOID(tx,ty+1,size)
	           voroid = tex1Dfetch(pbaTexColor,id);							//
	           id2    = TOID(voroid.x,voroid.y,size);						//
	    float  imp_u  = tex1Dfetch(pbaTexParam,id2);						//

			id -= size*2;																//TOID(tx,ty+1,size)
			voroid = tex1Dfetch(pbaTexColor, id);							//
			id2 = TOID(voroid.x, voroid.y, size);						//
		float  imp_u2 = tex1Dfetch(pbaTexParam, id2);						//

		float imp_dx  = fabsf(imp_r-imp);
		float imp_dx2 = fabsf(imp_r2 - imp);
	    float imp_dy  = fabsf(imp_u-imp);
		float imp_dy2 = fabsf(imp_u2 - imp);
	    float Imp     = max(imp_dy2,max(imp_dx2,max(imp_dx,imp_dy)));
	    Imp = min(Imp,fabsf(length-Imp));
		  float Imp_sal = Imp / d2;										//Compute salience metric

		  if (Imp_sal >= threshold && Imp > 1 && !(Imp < 5 && d2 < 1.5f))									//REMARK: exclude Imp==1 points since these are the boundary
			  {
				  output[Id] = 255;																//By filling only in-skeleton-values, we reduce memory access somehow (writing to output[] is expensive)

				  int id = blockIdx.x + blockIdx.y * gridDim.x;
				  if (Imp > output_skelmaxImp[id])
				  {
					  output_skelmaxImp[id] = Imp;
					  output_skelmaxPos[id] = make_short2(tx, ty);
				  }
			  }
		importanceDevice[Id] = Imp;//CHANGE2: 
	  //}
	}
}


__global__ void kernelTopology(unsigned char* output, short2* output_set, short xm, short ym, short xM, short yM, short size, int maxpts)
{
	const int tx = blockIdx.x * blockDim.x + threadIdx.x;
	const int ty = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char t[9];

	if (tx>xm && ty>ym && tx<xM-1 && ty<yM-1)									//careful not to index outside the image; take into account the template size too
	{
	   int    id = TOID(tx, ty, size);
	   unsigned char  p  = tex1Dfetch(pbaTexGray,id);							//get the skeleton pixel at tx,ty
	   if (p)																	//if the pixel isn't skeleton, nothing to do
	   {
	     unsigned char idx=0;
		   for(int j=ty-1;j<=ty+1;++j)											//read the template into t[] for easier use
		   {
		     int id = TOID(tx-1, j, size);
	       for(int i=0;i<=2;++i,++id,++idx)
		         t[idx] = tex1Dfetch(pbaTexGray,id);								//get the 3x3 template centered at the skel point tx,ty
		   }

		   for(unsigned char r=0;r<4;++r)											//try to match all rotations of a pattern:
		   {
		     const unsigned char* rr = topo_rot[r];
	       for(unsigned char p=0;p<topo_NPATTERNS;++p)							//try to match all patterns:
	       {
	         const unsigned char* pat = topo_patterns[p];
			     unsigned char j = (p==0)? 0 : 7;									//Speedup: for all patterns except 1st, check only last 3 entries, the first 6 are identical for all patterns
			     for(;j<9;++j)														//try to match rotated pattern vs actual pattern
			       if (pat[j]!=t[rr[j]]) break;										//this rotation failed
			     if (j<6) break;													//Speedup: if we have a mismatch on the 1st 6 pattern entries, then none of the patterns can match
																				//		   since all templates have the same first 6 entries.

			     if (j==9)															//this rotation succeeded: mark the pixel as a topology event and we're done
			     {
				     int crt_gc = atomicInc(&topo_gc,maxpts);						//REMARK: this serializes (compacts) all detected endpoints in one array.
						 output_set[crt_gc] = make_short2(tx,ty);						//To do this, we use an atomic read-increment-return on a global counter,
																				//which is guaranteed to give all threads unique consecutive indexes in the array.
			       output[id] = 1;													//Also create the topology image
				     return;
			     }
		     }
		   }
	   }
	}
	else																		//Last thread: add zero-marker to the output point-set, so the reader knows how many points are really in there
	if (tx==xM-1 && ty==yM-1)													//Also reset the global vector counter topo_gc, for the next parallel-run of this function
	{
		//!!!topo_gc_last = topo_gc; topo_gc = 0;
	}									//We do this in the last thread so that no one modifies topo_gc from now on.
																				//REMARK: this seems to be the only way I can read a __device__ variable back to the CPU
}




void skelft2DParams(int floodBand_, int maurerBand_, int colorBand_)		//Set up some params of the FT algorithm
{
  floodBand   = floodBand_;
  maurerBand  = maurerBand_;
  colorBand   = colorBand_;
}





// Compute 2D FT / Voronoi diagram of a set of sites
// siteParam:   Site parameterization. 0 = non-site points; >0 = site parameter value.
// output:		FT. The (x,y) at (i,j) are the coords of the closest site to (i,j)
// size:        Texture size (pow 2)
void skelft2DFT(short* output, float* siteParam, short xm, short ym, short xM, short yM, int size)
{
    skelft2DInitializeInput(siteParam,size);								    // Initialization of already-allocated data structures

    skel2DFTCompute(xm, ym, xM, yM, floodBand, maurerBand, colorBand);			// Compute FT

    // Copy FT to CPU, if required
    if (output) hipMemcpy(output, pbaTextures[1], size*size*sizeof(short2), hipMemcpyDeviceToHost);
}








void skelft2DDT(short* outputDT, float threshold,								//Compute (thresholded) DT (into pbaTextures[2]) from resident FT (in pbaTextures[1])
					  short xm, short ym, short xM, short yM)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    hipBindTexture(0, pbaTexColor, pbaTextures[1]);							//Used to read the FT from

	if (threshold>=0)
	{
	  xm -= threshold; if (xm<0) xm=0;
	  ym -= threshold; if (ym<0) ym=0;
	  xM += threshold; if (xM>pbaTexSize-1) xM=pbaTexSize-1;
	  yM += threshold; if (yM>pbaTexSize-1) yM=pbaTexSize-1;

      kernelThresholdDT<<< grid, block >>>((unsigned char*)pbaTextures[2], pbaTexSize, threshold*threshold, xm-1, ym-1, xM+1, yM+1);
      hipUnbindTexture(pbaTexColor);

	  //Copy thresholded image to CPU
	  if (outputDT) hipMemcpy(outputDT, (unsigned char*)pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	}
	else
	{
	  xm = ym = 0; xM = yM = pbaTexSize-1;
	  kernelDT <<< grid, block >>>((short*)pbaTextures[2], pbaTexSize, threshold*threshold, xm-1, ym-1, xM+1, yM+1);
      hipUnbindTexture(pbaTexColor);
	  //Copy thresholded image to CPU
	  if (outputDT) hipMemcpy(outputDT, pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(short), hipMemcpyDeviceToHost);
	}
}



void skelft2DSkeleton(unsigned char* outputSkel, float length, float threshold,	//Compute thresholded skeleton (into pbaTextures[3]) from resident FT (in pbaTextures[1])
					  short xm,short ym,short xM,short yM,
					  float* globalMaxImp, short2* globalMaxPos, float* importanceHost, int fboSize)
{																				//length:     boundary length
	dim3 block = dim3(BLOCKX,BLOCKY);											//threshold:  skeleton importance min-value (below this, we ignore branches)
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);
	int  SZ	   = grid.x*grid.y;

	hipBindTexture(0, pbaTexColor, pbaTextures[1]);							//Used to read the resident FT
	hipBindTexture(0, pbaTexParam, pbaTexSiteParam);							//Used to read the resident boundary parameterization
	hipMemset(outputSkel,0,sizeof(unsigned char)*pbaTexSize*pbaTexSize);		//Faster to zero result and then fill only 1-values (see kernel)
	hipMemset(pbaTextures[0],0,sizeof(float)*SZ);								//Use pbaTextures[0] and [2] to collect the max-importance and argmax-importance per grid-block, respectively

	float* importanceDevice;
	hipMalloc((void**)&importanceDevice, fboSize * fboSize * sizeof(float));//CHANGE1: added importance buffer

	kernelSkel<<< grid, block >>>(outputSkel, xm, ym, xM-1, yM-1, pbaTexSize, threshold, length, (float*)pbaTextures[0],pbaTextures[2], SKELFT_FAR_AWAY, importanceDevice);
																				//Compute the skeleton; it's returned both as a texture (outputSkel)
	hipUnbindTexture(pbaTexColor);
	hipUnbindTexture(pbaTexParam);

	/*float* maxImp = new float[SZ];															//Now compute the global (per-image) importance max and arg-max.
	short2* maxPos = new short2[SZ];
	hipMemcpy(maxImp,pbaTextures[0],SZ*sizeof(float),hipMemcpyDeviceToHost);	//Get the per-grid-bloc max and arg-max from CUDA. We need this on the CPU to compute the per-image max and arg-max
	hipMemcpy(maxPos,pbaTextures[2],SZ*sizeof(short2),hipMemcpyDeviceToHost);	//*/

	hipMemcpy(importanceHost, importanceDevice, fboSize * fboSize * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(importanceDevice);

	//*globalMaxImp = -1;															//This will store the per-image max and arg-max of the skel importance.
	//for(int i=0;i<SZ;++i)														//Aggregate per-grid max values into the global max
	//   if (maxImp[i] > *globalMaxImp)
	 //  {
	//	   *globalMaxImp = maxImp[i];
	//	   *globalMaxPos = maxPos[i];
	 //  }
	//delete[] maxImp;
	//delete[] maxPos;
}




void skelft2DTopology(unsigned char* outputTopo, unsigned char* inputSkel, int* npts, short* outputPoints, //Compute topology-points of the resident skeleton (in pbaTextures[3])
					  short xm,short ym,short xM,short yM)
{
  int maxpts = (npts)? *npts : pbaTexSize*pbaTexSize;							//This is the max # topo-points we are going to return in outputPoints[]

	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

  hipBindTexture(0, pbaTexGray, inputSkel);								//Used to read the resident skeleton
	hipMemset(pbaTextures[4],0,sizeof(unsigned char)*pbaTexSize*pbaTexSize);	//Faster to zero result and then fill only 1-values (see kernel)

  unsigned int zero = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(topo_gc),&zero,sizeof(unsigned int),0,hipMemcpyHostToDevice);		//Set topo_gc to 0

  kernelTopology<<< grid, block >>>((unsigned char*)pbaTextures[4], pbaTextures[5], xm, ym, xM, yM, pbaTexSize, maxpts+1);

  hipUnbindTexture(pbaTexGray);

	if (outputPoints && maxpts)													//If output-point vector desired, copy the end-points, put in pbaTexture[5] as a vector of short2's,
	{																			//into caller space. We copy only 'maxpts' elements, as the user instructed us.
	  unsigned int num_pts;
		hipMemcpyFromSymbol(&num_pts,HIP_SYMBOL(topo_gc),sizeof(unsigned int),0,hipMemcpyDeviceToHost);		//Get #topo-points we have detected from the device-var from CUDA

		if (npts && num_pts)																			//Copy the topo-points to caller
		   hipMemcpy(outputPoints,pbaTextures[5],num_pts*sizeof(short2),hipMemcpyDeviceToHost);
		if (npts) *npts = num_pts;												//Return #detected topo-points to caller
	}

	if (outputTopo)																//If topology image desired, copy it into user space
		hipMemcpy(outputTopo,pbaTextures[4],pbaTexSize*pbaTexSize*sizeof(unsigned char), hipMemcpyDeviceToHost);
}




__global__ void kernelSiteFromSkeleton(short2* outputSites, int size)						//Initialize the Voronoi textures from the sites' encoding texture (parameterization)
{																							//REMARK: we interpret 'inputVoro' as a 2D texture, as it's much easier/faster like this
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx<size && ty<size)																	//Careful not to go outside the image..
	{
	  int i = TOID(tx,ty,size);
	  unsigned char param = tex1Dfetch(pbaTexGray,i);										//The sites-param has non-zero (parameter) values precisely on non-boundary points

	  short2& v = outputSites[i];
	  v.x = v.y = MARKER;																	//Non-boundary points are marked as 0 in the parameterization. Here we will compute the FT.
	  if (param)																			//These are points which define the 'sites' to compute the FT/skeleton (thus, have FT==identity)
	  {																						//We could use an if-then-else here, but it's faster with an if-then
	     v.x = tx; v.y = ty;
	  }
	}
}




__global__ void kernelSkelInterpolate(float* output, int size)
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx<size && ty<size)																	//Careful not to go outside the image..
	{
  	  int    id     = TOID(tx, ty, size);
	  short2 vid    = tex1Dfetch(pbaTexColor,id);
	  float  T      = sqrtf((tx-vid.x)*(tx-vid.x)+(ty-vid.y)*(ty-vid.y));
	  short2 vid2   = tex1Dfetch(pbaTexColor2,id);
	  float  D      = sqrtf((tx-vid2.x)*(tx-vid2.x)+(ty-vid2.y)*(ty-vid2.y));
	  float  B      = ((D)? min(T/2/D,0.5f):0.5) + 0.5*((T)? max(1-D/T,0.0f):0);
	  output[id]    = B;
	}
}




void skel2DSkeletonDT(float* outputSkelDT,short xm,short ym,short xM,short yM)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    hipBindTexture(0,pbaTexGray,pbaTextures[3]);							//Used to read the resident binary skeleton
    kernelSiteFromSkeleton<<<grid,block>>>(pbaTextures[0],pbaTexSize);		//1. Init pbaTextures[0] with sites on skeleton i.e. from pbaTexGray
	hipUnbindTexture(pbaTexGray);

	//!!Must first save pbaTextures[1] since we may need it later..
	hipMemcpy(pbaTextures[5],pbaTextures[1],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);
    skel2DFTCompute(xm, ym, xM, yM, floodBand, maurerBand, colorBand);		//2. Compute FT of the skeleton into pbaTextures[6]
    hipMemcpy(pbaTextures[6],pbaTextures[1],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);
    hipMemcpy(pbaTextures[1],pbaTextures[5],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);

	//Compute interpolation
    hipBindTexture(0,pbaTexColor,pbaTextures[1]);							// FT of boundary
    hipBindTexture(0,pbaTexColor2,pbaTextures[6]);							// FT of skeleton
	kernelSkelInterpolate<<<grid,block>>>((float*)pbaTextures[0],pbaTexSize);
	hipUnbindTexture(pbaTexColor);
	hipUnbindTexture(pbaTexColor2);
	if (outputSkelDT) hipMemcpy(outputSkelDT, pbaTextures[0], pbaTexSize * pbaTexSize * sizeof(float), hipMemcpyDeviceToHost);
}




__device__  bool fill_gc;														//Indicates if a fill-sweep did fill anything or not


__global__ void kernelFill(unsigned char* output, int size, unsigned char bg, unsigned char fg, short xm, short ym, short xM, short yM, bool ne)
{																				//Fill image in pbaTexGray[] from a preset seed; result goes into output[] image.
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if (tx>xm && ty>ym && tx<xM && ty<yM)										//careful not to index outside the image..
	{
	  int    id0 = TOID(tx, ty, size);
	  unsigned char val = tex1Dfetch(pbaTexGray,id0);							//
	  if (val==fg)																//do we have a filled pixel? Then fill all to left/top/up/bottom of it which is background
	  {
	    bool fill = false;
		int id = id0;
		if (ne)																	//fill in north+east direction:
		{
			for(short x=tx+1;x<xM;++x)											//REMARK: here and below, the interesting thing is that it's faster, by about 10-15%, to fill a whole
			{																	//        scanline rather than only until the current block's borders (+1). The reason is that filling a whole
																				//		  scanline decreases the total #sweeps, which seems to be the limiting speed factor
			  if (tex1Dfetch(pbaTexGray,++id)!=bg) break;
			  output[id] = fg; fill = true;
			}

			id = id0;
			for(short y=ty-1;y>ym;--y)
			{
			  if (tex1Dfetch(pbaTexGray,id-=size)!=bg) break;
			  output[id] = fg; fill = true;
			}
		}
		else																	//fill in south+west direction:
		{
			for(short x=tx-1;x>xm;--x)
			{
			  if (tex1Dfetch(pbaTexGray,--id)!=bg) break;
			  output[id] = fg; fill = true;
			}

			id = id0;
			for(short y=ty+1;y<yM;++y)
			{
			  if (tex1Dfetch(pbaTexGray,id+=size)!=bg) break;
			  output[id] = fg; fill = true;
			}
		}

	    if (fill) fill_gc = true;												//if we filled anything, inform caller; we 'gather' this info from a local var into the
																				//global var here, since it's faster than writing the global var in the for loops
	  }
    }
}




__global__ void kernelComputeCoreComponent(unsigned char* output, int size, short2* output_set, int maxpts)
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if (tx>=0 && ty>=0 && tx<size && ty<size)									//careful not to index outside the image..
	{
  	  int            id = TOID(tx, ty, size);
	  unsigned char val = tex1Dfetch(pbaTexGray,id);							//
	  if (val==255)																//Skeleton component _not_ in the core-skeleton: Erase it
	     output[id] = 0;
	  else if (val==128)														//Skeleton core component: keep this, and mark it 255 so we're clear what it is
	  {
	     output[id] = 255;
		 int crt_gc = atomicInc(&topo_gc,maxpts);								//REMARK: this serializes (compacts) all detected endpoints in one array.
		 output_set[crt_gc] = make_short2(tx,ty);								//To do this, we use an atomic read-increment-return on a global counter
	  }
	}
																				//Last thread: add zero-marker to the output point-set, so the reader knows how many points are really in there
	if (tx==size-1 && ty==size-1)												//Also reset the global vector counter topo_gc, for the next parallel-run of this function
	{
		topo_gc_last = topo_gc; topo_gc = 0;									//We do this in the last thread so that no one modifies topo_gc from now on.

	}																			//REMARK: this seems to be the only way I can read a __device__ variable back to the CPU
}



int skelft2DFill(unsigned char* outputFill, short sx, short sy, short xm, short ym, short xM, short yM, unsigned char fill_value)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    unsigned char background;
	int id = sy * pbaTexSize + sx;
	hipMemcpy(&background,(unsigned char*)pbaTextures[2]+id,sizeof(unsigned char),hipMemcpyDeviceToHost); //See which is the value we have to fill from (sx,sy)

	hipMemset(((unsigned char*)pbaTextures[2])+id,fill_value,sizeof(unsigned char));					//Fill the seed (x,y) on the GPU

	hipBindTexture(0, pbaTexGray, pbaTextures[2]);														//Used to read the thresholded DT

	int iter=0;
	bool xy = true;																						//Direction of filling for current sweep: either north-east or south-west
																										//This kind of balances the memory-accesses nicely over kernel calls
	for(;;++iter,xy=!xy)																				//Keep filling a sweep at a time until we have no background pixels anymore
	{
	   bool filled = false;																				//Initialize flag: we didn't fill anything in this sweep
	   hipMemcpyToSymbol(HIP_SYMBOL(fill_gc),&filled,sizeof(bool),0,hipMemcpyHostToDevice);						//Pass flag to CUDA
       kernelFill<<<grid, block>>>((unsigned char*)pbaTextures[2],pbaTexSize,background,fill_value,xm,ym,xM,yM,xy);
																										//One fill sweep
	   hipMemcpyFromSymbol(&filled,HIP_SYMBOL(fill_gc),sizeof(bool),0,hipMemcpyDeviceToHost);						//See if we filled anything in this sweep
	   if (!filled) break;																				//Nothing filled? Then we're done, the image didn't change
	}
	hipUnbindTexture(pbaTexGray);

	if (outputFill) hipMemcpy(outputFill, (unsigned char*)pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

	return iter;																						//Return #iterations done for the fill - useful as a performance measure for caller
}



void skelft2DComputeCoreComponent(unsigned char* outputFill, int* npts, short2* outputPoints)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    int maxpts = (npts)? *npts : pbaTexSize*pbaTexSize;												//This is the max # skel-points we are going to return in outputPoints[]

    hipBindTexture(0, pbaTexGray, outputFill);														//Used to read the input data

    unsigned int zero = 0;
  	hipMemcpyToSymbol(HIP_SYMBOL(topo_gc),&zero,sizeof(unsigned int),0,hipMemcpyHostToDevice);				//Set topo_gc to 0

    kernelComputeCoreComponent<<<grid, block>>>(outputFill, pbaTexSize, pbaTextures[5], maxpts+1);

    hipUnbindTexture(pbaTexGray);

	if (outputPoints && maxpts)													//If output-point vector desired, copy the end-points, put in pbaTexture[5] as a vector of short2's,
	{																			//into caller space. We copy only 'maxpts' elements, as the user instructed us.
	  unsigned int num_pts;
		hipMemcpyFromSymbol(&num_pts,HIP_SYMBOL(topo_gc_last),sizeof(unsigned int),0,hipMemcpyDeviceToHost);		//Get #points we have detected from the device-var from CUDA
		if (npts && num_pts)																			//Copy the topo-points to caller
		   hipMemcpy(outputPoints,pbaTextures[5],num_pts*sizeof(short2),hipMemcpyDeviceToHost);
		if (npts) *npts = num_pts;												//Return #detected topo-points to caller
	}
}
